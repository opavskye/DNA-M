/* Copyright 2012 by Erik Opavsky
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

void printDeviceSequences (char * d_sequences, int numSequences, int sequenceLength) {
  char * temp = (char *) malloc (sizeof (char) * numSequences * sequenceLength);

  hipMemcpy (temp, d_sequences, sizeof (char) * numSequences * sequenceLength, hipMemcpyDeviceToHost);

  // for (int i = 0; i < numSequences * sequenceLength; i += sequenceLength)
  //  printf ("d_sequences[%d] = %s\n", i / sequenceLength, temp + i);  
  for (int i = 0; i < numSequences; i++) {
    printf ("d_sequences[%d] = ", i);
    for (int j = 0; j < sequenceLength; j++)
      printf ("%c", *(temp + i * sequenceLength + j));  
    printf ("\n");
  }

  free (temp);
}

void printFirstLastBuckets (char * d_bucketSequence, int numBuckets, int matchLength, int sequenceLength) {
  char * temp = (char *) malloc (sizeof (char) * sequenceLength);

  hipMemcpy (temp, d_bucketSequence, sizeof (char) * sequenceLength, hipMemcpyDeviceToHost);
  // cudaMemcpy (temp2, (d_bucketSequence + numBuckets * sizeof (char)), sizeof (char) * matchLength, cudaMemcpyDeviceToHost);

  printf ("first bucket = ");
  for (int i = 0; i < matchLength; i++)
    printf("%c", *(temp + i));

  printf("\nlast bucket = ");
  for (int i = 0; i < matchLength; i++)
    printf("%c", *(temp + numBuckets - 1 + i));
  printf("\n");
  //  printf("numbuckets = %d\n", numBuckets);
  free (temp);
  // free (temp2);
}

void printDeviceFirstLast (char * d_sequences, int numSequences, int sequenceLength) {
  char * temp = (char *) malloc (sizeof (char) * 2 * sequenceLength);

  hipMemcpy (temp, d_sequences, sizeof (char) * sequenceLength, hipMemcpyDeviceToHost);
  hipMemcpy (temp + sequenceLength, d_sequences + sequenceLength * (numSequences - 1), sizeof (char) * sequenceLength, hipMemcpyDeviceToHost);

  int i;

  printf ("d_sequences[0] = ");
  for (i = 0; i < sequenceLength; i++)
    printf ("%c", *(temp + i));
  printf ("\n");

  printf ("d_sequences[%d] = ", numSequences - 1);
  for (; i < sequenceLength * 2; i++)
    printf ("%c", *(temp + i));
  printf ("\n");

  free (temp);
}


void printFirstLast (char ** sequences, int numSequences, int sequenceLength) {
  printf("sequences[0] = %s\n", sequences[0]);
  printf("sequences[%d] = %s\n", numSequences - 1, sequences[numSequences - 1]);
}

void printSequences (char ** sequences, int numSequences, int sequenceLength) {
  for (int i = 0; i < numSequences; i++) {
    printf ("sequences[%d] = ", i);
    for (int j = 0; j < sequenceLength; j++)
      printf ("%c", sequences[i][j]);
    printf ("\n");
  }
}

/*
  void printFlatSequences (char * sequences, int numSequences, int sequenceLength) {
  for (int i = 0; i < numSequences; i++)
  printf ("flat_sequences[%d] = %s\n", i , sequences + i * sequenceLength);  
  }
*/
