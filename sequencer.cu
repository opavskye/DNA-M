#include "hip/hip_runtime.h"
#include "printFunctions.cu"
#include <time.h>

#define THREADS_PER_BLOCK 1024

char * copySequenceToDevice (char ** sequences, int numSequences, int sequenceLength) {
  char * d_sequences;
  hipMalloc (&d_sequences, sizeof (char) * sequenceLength * numSequences);

  for (int i = 0; i < numSequences; i++)
    hipMemcpy (d_sequences + i * sequenceLength, *(sequences + i), sizeof (char) * sequenceLength, hipMemcpyHostToDevice);

  return d_sequences;
}

__global__ void createBuckets (char * sequence, char * buckets, int sequenceLength, int numBuckets, int matchLength, int bucketsPerThread) {
	  
  int index = threadIdx.x + (blockIdx.x % bucketsPerThread * blockDim.x);
 
  if (index < numBuckets)
    for (int i = 0; i < matchLength; i++)
      *(buckets + matchLength * index + i) = *(sequence + index + i);
}

__global__ void assignBuckets (char * sequences, char * buckets, uint * bucketCounts, int numSequences, int sequenceLength, int numBuckets, int matchLength, double matchAccuracy, int bucketsPerThread) {
	  
	  
  // read buckets into shared memory for faster access
  extern __shared__ char sharedSequence[];
  for (int i = threadIdx.x; i < sequenceLength; i += blockDim.x)
    sharedSequence[i] = sequences[blockIdx.x * sequenceLength + i];
  
  // if (threadIdx.x == 0 && blockIdx.x == 1)
  //  printf ("shared sequence == %s\n", sharedSequence);

  syncthreads();
  
  int numMatches = 0;
  int bucketIndex;

  for (int k = 1; k <= bucketsPerThread; k++) {
    for (int i = 0; i < numBuckets; i++) {
      for (int j = 0; j < matchLength; j++) {
        if ((bucketIndex = threadIdx.x * k * matchLength + j) < numBuckets * matchLength) {
          if (*(buckets + bucketIndex) == *(sharedSequence + i + j))
            numMatches++;

          if (numMatches / (double) matchLength >= matchAccuracy) {
            atomicInc (bucketCounts + threadIdx.x * k, UINT_MAX);
            // return;
          }

        }
      }  

      numMatches = 0;
    }
  }
  
  // atomicInc (bucketCounts + numBuckets, UINT_MAX);
}

uint * sequencer (char * d_sequences, int numSequences, int sequenceLength, int matchLength, double matchAccuracy) {

  // printSequences (sequences, numSequences, sequenceLength);
  // printDeviceSequences (d_sequences, numSequences, sequenceLength);

  // choose a random sequence to create buckets from
  srand (time (NULL));
  int bucketSequence = 0;//rand() % numSequences;
	  
  // printf ("bucketSequence = %d\n", bucketSequence);

  // create the buckets
  char * d_buckets;
  int numBuckets = sequenceLength - matchLength + 1;
  hipMalloc (&d_buckets, sizeof (char) * numBuckets * matchLength);

  int numThreads = THREADS_PER_BLOCK;
  int bucketsPerThread = ceil (numBuckets / (float) numThreads);

  if (numThreads > numBuckets)
    numThreads = numBuckets;

  createBuckets<<<bucketsPerThread, numThreads>>> (d_sequences + bucketSequence * sequenceLength, d_buckets, numBuckets, sequenceLength, matchLength, bucketsPerThread);

  printDeviceFirstLast (d_sequences, numSequences, sequenceLength);
  printDeviceFirstLast (d_buckets, numBuckets, matchLength);

  // make counters for each bucket, with the last one counting how many didn't fit
  // into any buckets
  uint * d_bucketCounts;
  hipMalloc (&d_bucketCounts, sizeof (uint) * (numBuckets + 1));
  hipMemset (d_bucketCounts, 0, sizeof (uint) * (numBuckets + 1));

  // each block is a sequence
  // each thread assigns bucketsPerThread number of buckets
  assignBuckets<<<numSequences, numThreads, sizeof (char) * sequenceLength>>> (d_sequences, d_buckets, d_bucketCounts, numSequences, sequenceLength, numBuckets, matchLength, matchAccuracy, bucketsPerThread);

  hipDeviceSynchronize();

  uint * bucketCounts = (uint *) malloc (sizeof (uint) * (numBuckets + 1));
  hipMemcpy (bucketCounts, d_bucketCounts, sizeof (uint) * (numBuckets + 1), hipMemcpyDeviceToHost);

  // for (int i = 0; i < numBuckets + 1; i++)
  // printf ("bucketCounts[%d] = %u\n", i, *(bucketCounts + i));
  printf("\nnow printing after assignBuckets:\n");
  printDeviceFirstLast (d_sequences, numSequences, sequenceLength);
  printDeviceFirstLast (d_buckets, numBuckets, matchLength); 

  // printDeviceSequences (d_buckets, numBuckets, matchLength);

  // run kernel in loop from length of sequence down to ~10 or so to see
  // which bucket sizes give good results
  // will need an array which holds what the matching pattern is
  // will need an array to store data of which sequences have matching pattern
	  
  hipFree (d_bucketCounts);
  hipFree (d_buckets);

  return bucketCounts;
}

__global__ void counterKernel (char * sequences, int sequenceLength, char * query, int queryLength, uint * count, double matchAccuracy) {

  // read query into shared memory for faster access
  extern __shared__ char sharedQuery[];
  if (threadIdx.x < queryLength)
    *(sharedQuery + threadIdx.x) = query[threadIdx.x];

  int numMatches = 0;
  int startSpot = sequenceLength * blockIdx.x + threadIdx.x;

  for (int i = 0; i < queryLength; i++) {
    if (*(sequences + startSpot + i) == *(query + i))
      numMatches++;
  }

  if (numMatches / (double) queryLength >= matchAccuracy)
    atomicInc (count, UINT_MAX);
}


uint counter (char ** sequences, int numSequences, int sequenceLength, char * query, int queryLength, double matchAccuracy) {
	 
  // put sequences into device memory
  char * d_sequences = copySequenceToDevice (sequences, numSequences, sequenceLength);

  // put query into device memory
  char * d_query;
  hipMalloc (&d_query, queryLength * sizeof (char));
  hipMemcpy (d_query, query, queryLength * sizeof (char), hipMemcpyHostToDevice);

  // counts of how many times the query was found
  uint count = 0;
  uint * d_count;
  hipMalloc (&d_count, sizeof (uint));
  hipMemcpy (d_count, &count, sizeof (uint), hipMemcpyHostToDevice);

  counterKernel<<<numSequences, sequenceLength - queryLength + 1, queryLength * sizeof (char)>>> (d_sequences, sequenceLength, d_query, queryLength, d_count, matchAccuracy);

  hipMemcpy (&count, d_count, sizeof (uint), hipMemcpyDeviceToHost);

  hipFree (d_count);
  hipFree (d_query);
  hipFree (d_sequences);

  return count;
}
