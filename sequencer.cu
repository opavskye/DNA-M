#include "hip/hip_runtime.h"
#include "printFunctions.cu"
#include <time.h>

#define THREADS_PER_BLOCK 1024

char * copySequenceToDevice (char ** sequences, int numSequences, int sequenceLength) {
  char * d_sequences;
  hipMalloc (&d_sequences, sizeof (char) * sequenceLength * numSequences);

  for (int i = 0; i < numSequences; i++)
    hipMemcpy (d_sequences + i * sequenceLength, *(sequences + i), sizeof (char) * sequenceLength, hipMemcpyHostToDevice);

  return d_sequences;
}

__global__ void createBuckets (char * sequence, char * buckets, int sequenceLength, int numBuckets, int matchLength, int bucketsPerThread) {
	  
  int index = threadIdx.x + (blockIdx.x % bucketsPerThread * blockDim.x);
 
  if (index < numBuckets)
    for (int i = 0; i < matchLength; i++)
      *(buckets + matchLength * index + i) = *(sequence + index + i);
}

__global__ void assignBuckets (char * sequences, char * buckets, uint * bucketCounts, int numSequences, int sequenceLength, int numBuckets, int matchLength, double matchAccuracy, int bucketsPerThread) {
	  
	  
  // use shared memory for the sequence and buckets
  extern __shared__ char shared[];
  char * sharedSequence = &shared[0];
  char * sharedBuckets = &shared[sequenceLength];

  // fill sharedSequence
  for (int i = threadIdx.x; i < sequenceLength; i += blockDim.x)
    sharedSequence[i] = sequences[blockIdx.x * sequenceLength + i];
  
  // fill sharedBuckets
  for (int i = threadIdx.x; i < matchLength * numBuckets; i += blockDim.x)
    sharedBuckets[i] = buckets[i];
  

  // if (threadIdx.x == 0 && blockIdx.x == 1)
  // printf ("shared sequence == %s\n", sharedSequence);

  syncthreads();
  
  int numMatches = 0;
  int bucketIndex;

  for (int k = 0; k < bucketsPerThread; k++) {
    if ((bucketIndex = threadIdx.x + k * numBuckets / bucketsPerThread) < numBuckets) {
      for (int i = 0; i < numBuckets; i++) {
        for (int j = 0; j < matchLength; j++) {
          if (*(sharedBuckets + bucketIndex * matchLength + j) == *(sharedSequence + i + j))
            numMatches++;

          //printf("x\n");
          // if (numMatches / (double) matchLength >= matchAccuracy) {
          // atomicInc (bucketCounts + bucketIndex, UINT_MAX);
          // return;
          //  }
          atomicAdd (bucketCounts + bucketIndex, (numMatches / (double) matchLength >= matchAccuracy));

        }
        numMatches = 0;
      }  
    }
  }
  
  // atomicInc (bucketCounts + numBuckets, UINT_MAX);
}

uint * sequencer (char * d_sequences, int numSequences, int sequenceLength, int matchLength, double matchAccuracy) {

  // printSequences (sequences, numSequences, sequenceLength);
  // printDeviceSequences (d_sequences, numSequences, sequenceLength);

  // choose a random sequence to create buckets from
  srand (time (NULL));
  int bucketSequence = 0;//rand() % numSequences;
	  
  // printf ("bucketSequence = %d\n", bucketSequence);

  // create the buckets
  char * d_buckets;
  int numBuckets = sequenceLength - matchLength + 1;
  hipMalloc (&d_buckets, sizeof (char) * numBuckets * matchLength);

  int numThreads = THREADS_PER_BLOCK;
  int bucketsPerThread = ceil (numBuckets / (float) numThreads);

  if (numThreads > numBuckets)
    numThreads = numBuckets;

  createBuckets<<<bucketsPerThread, numThreads>>> (d_sequences + bucketSequence * sequenceLength, d_buckets, numBuckets, sequenceLength, matchLength, bucketsPerThread);

  // make counters for each bucket, with the last one counting how many didn't fit
  // into any buckets
  uint * d_bucketCounts;
  hipMalloc (&d_bucketCounts, sizeof (uint) * numBuckets);
  hipMemset (d_bucketCounts, 0, sizeof (uint) * numBuckets);

  /*
  printDeviceFirstLast (d_sequences, numSequences, sequenceLength);
  printDeviceFirstLast (d_buckets, numBuckets, matchLength);
  */
  // each block is a sequence
  // each thread assigns bucketsPerThread number of buckets
  assignBuckets<<<numSequences, numThreads, sizeof (char) * (matchLength * numBuckets + sequenceLength)>>> (d_sequences, d_buckets, d_bucketCounts, numSequences, sequenceLength, numBuckets, matchLength, matchAccuracy, bucketsPerThread);

  hipDeviceSynchronize();

  /*
  printf("\nnow printing after assignBuckets:\n\n");
  printDeviceFirstLast (d_sequences, numSequences, sequenceLength);
  printDeviceFirstLast (d_buckets, numBuckets, matchLength); 
  */
  uint * bucketCounts = (uint *) malloc (sizeof (uint) * numBuckets);
  hipMemcpy (bucketCounts, d_bucketCounts, sizeof (uint) * numBuckets, hipMemcpyDeviceToHost);

  // for (int i = 0; i < numBuckets + 1; i++)
  // printf ("bucketCounts[%d] = %u\n", i, *(bucketCounts + i));


  // printDeviceSequences (d_buckets, numBuckets, matchLength);

  // run kernel in loop from length of sequence down to ~10 or so to see
  // which bucket sizes give good results
  // will need an array which holds what the matching pattern is
  // will need an array to store data of which sequences have matching pattern
	  
  hipFree (d_bucketCounts);
  hipFree (d_buckets);

  return bucketCounts;
}

__global__ void counterKernel (char * sequences, int sequenceLength, char * query, int queryLength, uint * count, double matchAccuracy) {

  // read query into shared memory for faster access
  extern __shared__ char sharedQuery[];
  if (threadIdx.x < queryLength)
    *(sharedQuery + threadIdx.x) = query[threadIdx.x];

  int numMatches = 0;
  int startSpot = sequenceLength * blockIdx.x + threadIdx.x;

  for (int i = 0; i < queryLength; i++) {
    if (*(sequences + startSpot + i) == *(query + i))
      numMatches++;
  }

  if (numMatches / (double) queryLength >= matchAccuracy)
    atomicInc (count, UINT_MAX);
}


uint counter (char ** sequences, int numSequences, int sequenceLength, char * query, int queryLength, double matchAccuracy) {
	 
  // put sequences into device memory
  char * d_sequences = copySequenceToDevice (sequences, numSequences, sequenceLength);

  // put query into device memory
  char * d_query;
  hipMalloc (&d_query, queryLength * sizeof (char));
  hipMemcpy (d_query, query, queryLength * sizeof (char), hipMemcpyHostToDevice);

  // counts of how many times the query was found
  uint count = 0;
  uint * d_count;
  hipMalloc (&d_count, sizeof (uint));
  hipMemcpy (d_count, &count, sizeof (uint), hipMemcpyHostToDevice);

  counterKernel<<<numSequences, sequenceLength - queryLength + 1, queryLength * sizeof (char)>>> (d_sequences, sequenceLength, d_query, queryLength, d_count, matchAccuracy);

  hipMemcpy (&count, d_count, sizeof (uint), hipMemcpyDeviceToHost);

  hipFree (d_count);
  hipFree (d_query);
  hipFree (d_sequences);

  return count;
}
