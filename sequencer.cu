#include "hip/hip_runtime.h"
#include "printFunctions.cu"
#include <time.h>

#define THREADS_PER_BLOCK 1024

char * copySequenceToDevice (char ** sequences, int numSequences, int sequenceLength) {
  char * d_sequences;
  hipMalloc (&d_sequences, sizeof (char) * sequenceLength * numSequences);

  for (int i = 0; i < numSequences; i++)
    hipMemcpy (d_sequences + i * sequenceLength, *(sequences + i), sizeof (char) * sequenceLength, hipMemcpyHostToDevice); 

  return d_sequences;
}

__global__ void createBuckets (char * sequence, char * buckets, int sequenceLength, int matchLength) {
  
  int index = threadIdx.x + blockIdx.x * threadIdx.x;

  for (int i = 0; i < matchLength; i++)
    *(buckets + matchLength * index + i) = *(sequence + index + i);
}

__global__ void assignBuckets (char * sequences, char * buckets, uint * bucketCounts, int numSequences, int sequenceLength, int numBuckets, int matchLength, double matchAccuracy) {
  
  
  // read buckets into shared memory for faster access
  extern __shared__ char sharedBuckets[];
  for (int i = threadIdx.x; i < numBuckets * matchLength; i += blockDim.x)
    if (i < numBuckets)
      for (int j = 0; j < matchLength; j++)
        sharedBuckets[i * matchLength + j] = buckets[i * matchLength + j];
  

  int numMatches = 0;

  for (int i = 0; i < numBuckets; i++) {
    for (int j = 0; j < matchLength; j++) 
      if (*(sequences + blockIdx.x * sequenceLength + threadIdx.x + j) == *(sharedBuckets + i * matchLength + j))
        numMatches++;
    
    if (numMatches / (double) matchLength >= matchAccuracy)
      atomicInc (bucketCounts + i, UINT_MAX);

    numMatches = 0;
  }    

  atomicInc (bucketCounts + numBuckets, UINT_MAX);
}

void sequencer (char ** sequences, int numSequences, int sequenceLength, int matchLength, double matchAccuracy) {

  // put sequences into device memory
  char * d_sequences = copySequenceToDevice (sequences, numSequences, sequenceLength);

  // printSequences (sequences, numSequences, sequenceLength);
  // printDeviceSequences (d_sequences, numSequences, sequenceLength);

  // choose a random sequence to create buckets from
  srand (time (NULL));
  int bucketSequence = rand() % numSequences;
  
  // printf ("bucketSequence = %d\n", bucketSequence);

  // create the buckets
  char * d_buckets;
  int numBuckets = sequenceLength - matchLength + 1;
  hipMalloc (&d_buckets, sizeof (char) * numBuckets * matchLength); 

  int numThreads = THREADS_PER_BLOCK;
  int numBlocks = ceil (numBuckets / (float) numThreads);

  if (numThreads > numBuckets)
    numThreads = numBuckets;

  createBuckets<<<numBlocks, numThreads>>> (d_sequences + bucketSequence * sequenceLength, d_buckets, sequenceLength, matchLength);

  // make counters for each bucket, with the last one counting how many didn't fit
  //  into any buckets
  uint * d_bucketCounts;
  hipMalloc (&d_bucketCounts, sizeof (uint) * (numBuckets + 1));
  hipMemset (d_bucketCounts, 0, sizeof (uint) * (numBuckets + 1));

  // count how many sequences go into each bucket
  numThreads = numBuckets;
  numBlocks = numSequences;
  assignBuckets<<<numBlocks, numThreads, (sizeof (char) * numBuckets * matchLength)>>> (d_sequences, d_buckets, d_bucketCounts, numSequences, sequenceLength, numBuckets, matchLength, matchAccuracy);

  uint * bucketCounts = (uint *) malloc (sizeof (uint) * (numBuckets + 1));
  hipMemcpy (bucketCounts, d_bucketCounts, sizeof (uint) * (numBuckets + 1), hipMemcpyDeviceToHost);

  for (int i = 0; i < numBuckets + 1; i++)
    printf ("bucketCount[%d] = %u\n", i, *(bucketCounts + i));
  

  // printDeviceSequences (d_buckets, numBuckets, matchLength);

  // run kernel in loop from length of sequence down to ~10 or so to see
  //  which bucket sizes give good results
  //  will need an array which holds what the matching pattern is
  //  will need an array to store data of which sequences have matching pattern
  

  free (bucketCounts);
  hipFree (d_bucketCounts);
  hipFree (d_buckets);
  hipFree (d_sequences);

}
