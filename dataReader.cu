#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "sequencer.cu"

int readSequences (char * fileName, char ** sequences, int numSequences) {

  FILE *dataFile;
  if ((dataFile = fopen (fileName, "r")) == NULL) {
    printf("The file %s could not be opened.\n", fileName);
    return 0;
  }

  while (getc (dataFile) != ',');
  for (int i = 0; i < numSequences; i++) {

    // skip first column
    while (getc (dataFile) != ',');
    fscanf (dataFile, "%s", sequences[i]);

  }

  fclose (dataFile);
  return 1;
}


int main (int argc, char *argv[]) {
  
  char fileName[100];
  strcpy (fileName, "../data/");

  int numSequences = 125;
  int sequenceLength = 200;
  
  int matchLength = 5;
  double matchAccuracy = .8;

  // allocate memory for sequences
  char ** sequences =  sequences = (char **) malloc (numSequences * sizeof (char *));
  for (int i = 0; i < numSequences; i++)
    *(sequences + i) = (char *) malloc ((sequenceLength + 1) * sizeof (char));


  if (argc >= 1) 
    { 
      // append argv[1] to the end of path to data folder
      int i = 0; 
      while ((fileName[(i++) + 8] = argv[1][i]) != '\0');
    } 
  else
    {
      printf ("Please run again with a filename input from the data folder.\n");
      return 1;
    }

  // read in the data
  if (!readSequences (fileName, sequences, numSequences))
    printf ("error reading data\n");

  // printSequences (sequences, numSequences);

  sequencer (sequences, numSequences, sequenceLength, matchLength, matchAccuracy);

  // free all allocated memory
  for (int i = 0; i < numSequences; i++)
    free (sequences[i]);
  free (sequences);

  return 0;
}
