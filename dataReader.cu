#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "sequencer.cu"

int readSequences (char * fileName, char ** sequences, int numSequences) {

  FILE *dataFile;
  if ((dataFile = fopen (fileName, "r")) == NULL) {
    printf("The file %s could not be opened.\n", fileName);
    return 0;
  }

  while (getc (dataFile) != ',');
  for (int i = 0; i < numSequences; i++) {

    // skip first column
    while (getc (dataFile) != ',');
    fscanf (dataFile, "%s", sequences[i]);

  }

  fclose (dataFile);
  return 1;
}


int main (int argc, char *argv[]) {
  
  char fileName[100];
  strcpy (fileName, "../data/");

  char fileEnd[100] = "sample.csv";
  int numSequences = 125;
  int sequenceLength = 200;
  int matchLength = 10;
  double matchAccuracy = .8;

  if (argc < 2) {
    printf ("Please enter the name of the data file: ");
    scanf ("%s", fileEnd);

    printf ("Please enter the number of sequences: ");
    scanf ("%d", &numSequences);

    printf ("Please enter the length of the sequences: ");
    scanf ("%d", &sequenceLength);

    printf ("Please enter the length of the matching substrings: ");
    scanf ("%d", &matchLength);

    printf ("Please enter the minimum accuracy of the matches: ");
    scanf ("%lf", &matchAccuracy);
  }


  // allocate memory for sequences
  char ** sequences =  sequences = (char **) malloc (numSequences * sizeof (char *));
  for (int i = 0; i < numSequences; i++)
    *(sequences + i) = (char *) malloc ((sequenceLength + 1) * sizeof (char));

  // append fileEnd to the end of path to data folder
  int i = 0; 
  while ((fileName[(i++) + 8] = fileEnd[i]) != '\0');

  // read in the data
  if (!readSequences (fileName, sequences, numSequences))
    printf ("error reading data\n");

  // printSequences (sequences, numSequences);

  // sequencer (sequences, numSequences, sequenceLength, matchLength, matchAccuracy);
  char * s1 = "AGAGTTGTGG";
  char * s2 = "CAGGCAGCTC";
  char * s3 = "CTAACTGGGG";

  printf ("counter 1 = %u\n", counter (sequences, numSequences, sequenceLength, s1, 10, .8));
  printf ("counter 2 = %u\n", counter (sequences, numSequences, sequenceLength, s2, 10, .8));
  printf ("counter 3 = %u\n", counter (sequences, numSequences, sequenceLength, s3, 10, .8));

  // free all allocated memory
  for (int i = 0; i < numSequences; i++)
    free (sequences[i]);
  free (sequences);

  return 0;
}
